#include <hip/hip_runtime.h>

#include <time.h>

#define N 32*1024*1024
#define BLOCK_SIZE 256

template <unsigned int blockSize>
__device__ void warpReduce(volatile float* cache,int tid){
    if(blockSize >= 64)cache[tid]+=cache[tid+32];
    if(blockSize >= 32)cache[tid]+=cache[tid+16];
    if(blockSize >= 16)cache[tid]+=cache[tid+8];
    if(blockSize >= 8)cache[tid]+=cache[tid+4];
    if(blockSize >= 4)cache[tid]+=cache[tid+2];
    if(blockSize >= 2)cache[tid]+=cache[tid+1];
}

template <unsigned int blockSize, int NUM_PER_THREAD>
__global__ void reduce_v6(float *g_idata,float *g_odata){
    __shared__ float sdata[BLOCK_SIZE];

    // each thread loads one element from global to shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*(blockDim.x * NUM_PER_THREAD) + threadIdx.x;
    sdata[tid] = 0;
    #pragma unroll
    for(int iter=0; iter<NUM_PER_THREAD; iter++){
        sdata[tid] += g_idata[i+iter*blockSize];
    }
    __syncthreads();

    // do reduction in shared mem
    if(blockSize>=512){
        if(tid<256){
            sdata[tid]+=sdata[tid+256];
        }
        __syncthreads();
    }
    if(blockSize>=256){
        if(tid<128){
            sdata[tid]+=sdata[tid+128];
        }
        __syncthreads();
    }
    if(blockSize>=128){
        if(tid<64){
            sdata[tid]+=sdata[tid+64];
        }
        __syncthreads();
    }
    
    // write result for this block to global mem
    if(tid<32)warpReduce<blockSize>(sdata,tid);
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

int main() {
    float *input_host = (float*)malloc(N*sizeof(float));
    float *input_device;
    hipMalloc((void **)&input_device, N*sizeof(float));
    for (int i = 0; i < N; i++) input_host[i] = 2.0;
    hipMemcpy(input_device, input_host, N*sizeof(float), hipMemcpyHostToDevice);

    const int block_num = 1024;
    const int NUM_PER_BLOCK = N / block_num;
    const int NUM_PER_THREAD = NUM_PER_BLOCK / BLOCK_SIZE;
    float *output_host = (float*)malloc((block_num) * sizeof(float));
    float *output_device;
    hipMalloc((void **)&output_device, (block_num) * sizeof(float));
    
    dim3 grid(block_num, 1);
    dim3 block(BLOCK_SIZE, 1);
    reduce_v6<BLOCK_SIZE ,NUM_PER_THREAD><<<grid, block>>>(input_device, output_device);
    hipMemcpy(output_device, output_host, block_num * sizeof(float), hipMemcpyDeviceToHost);
    return 0;
}


